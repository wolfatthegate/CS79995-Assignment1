
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;

#define SIZE 64
#define THREADS 32 

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

__global__ void MatrixMul(const int *Md, const int *Nd, int *Pd, int Width) {

	// Calculate the row index of the Pd element and M
	int row = (blockIdx.y * blockDim.y) + threadIdx.y;

	// Calculate the column idenx of Pd and N
	int col = (blockIdx.x * blockDim.x) + threadIdx.x;

	if((row < Width) && (col < Width)) {
	// each thread computes one element of the block sub-matrix
		
		for (int k = 0; k < Width; ++k) {
		 	 // dot product or corresponding row and column. 
		 	 Pd[(row * Width) + col] += Md[(row * Width) + k] * Nd[(k * Width) + col];
		}
	}
}

// Check result on the CPU (single threaded)

void verify_result(vector<int> &a, vector<int> &b, vector<int> &c, int N) {
  	// Loop over every row...
  	float time;
	hipEvent_t start, stop;

	// start tracking the time
    hipEventCreate(&start) ;
	hipEventCreate(&stop) ;
	hipEventRecord(start, 0) ;

  	for (int i = 0; i < N; i++) {
    	// Loop every column...
    	for (int j = 0; j < N; j++) {
      		// For every element in the row-column pair
      		
      		int tmp = 0;
      		for (int k = 0; k < N; k++) {
        		// Accumulate the partial results
        	tmp += a[i * N + k] * b[k * N + j];
      		}

      		// Check against the CPU result
      		assert(tmp == c[i * N + j]);
    	}
  	}

    hipEventRecord(stop, 0) ;
	hipEventSynchronize(stop) ;
	hipEventElapsedTime(&time, start, stop) ;

	printf("Compute time on CPU:  %3.6f ms \n", time);
}

int main(void){
	
	// initialize event creation for time tracking
	float time;
	hipEvent_t start, stop;


	// Matrix size of 32 x 32; 
	int N = SIZE; 

	printf("Matrix Size: %d x %d\n", N, N); 

	// size (in bytes) of matrix
	size_t size = N * N * sizeof(int); 

	vector<int> host_a(N * N);
	vector<int> host_b(N * N); 
	vector<int> host_c(N * N); 

	// generate random indices between 0 and 1. 
	generate(host_a.begin(), host_a.end(), []() {return rand() % 2; });  
	generate(host_b.begin(), host_b.end(), []() {return rand() % 2; });

	// device memory allocation
	int *dev_a, *dev_b, *dev_c;

    hipMalloc(&dev_a, size);
    hipMalloc(&dev_b, size);
    hipMalloc(&dev_c, size);

    //copy data from host to device
    hipMemcpy(dev_a, host_a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b.data(), size, hipMemcpyHostToDevice);

    int BLOCKS = N/THREADS; 

    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS); 

    // start tracking the time
    hipEventCreate(&start) ;
	hipEventCreate(&stop) ;
	hipEventRecord(start, 0) ;

    // launch kernal

    printf("No. of blocks: %d x %d\n", BLOCKS, BLOCKS); 
    printf("No. of therads: %d x %d\n", THREADS, THREADS); 
    MatrixMul<<<blocks, threads>>> (dev_a, dev_b, dev_c, N); 

    // stop tracking the time
    hipEventRecord(stop, 0) ;
	hipEventSynchronize(stop) ;
	hipEventElapsedTime(&time, start, stop) ;


    hipMemcpy(host_c.data(), dev_c, size, hipMemcpyDeviceToHost);

	printf("Compute time on GPU:  %3.6f ms \n", time);

	// verify result on CPU
    verify_result(host_a, host_b, host_c, N);

    //free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

	return 0; 
}

